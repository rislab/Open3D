#include "open3d/core/Blob.h"
#include "open3d/core/CUDAUtils.h"
#include "open3d/core/ParallelFor.h"
#include "open3d/core/linalg/BlasWrapper.h"
#include "open3d/core/linalg/LapackWrapper.h"
#include "open3d/core/linalg/LinalgUtils.h"
#include "open3d/core/linalg/SolveLLTBatched.h"

namespace open3d {
namespace core {

void SolveCUDALLTBatched(void* A_data,
                         void* B_data,
                         int64_t batch_size,
                         int64_t cols,
                         Dtype dtype,
                         const Device& device) {
    hipsolverHandle_t handle = CuSolverContext::GetInstance().GetHandle(device);
    DISPATCH_LINALG_DTYPE_TO_TEMPLATE(dtype, [&]() {
        // prepare data
        scalar_t* A_ptr = static_cast<scalar_t*>(A_data);

        scalar_t** A_array = static_cast<scalar_t**>(
                MemoryManager::Malloc(batch_size * sizeof(scalar_t*), device));
        ParallelFor(device, batch_size,
                    [=] OPEN3D_DEVICE(int64_t workload_idx) {
                        A_array[workload_idx] =
                                A_ptr + (workload_idx * cols * cols);
                    });

        // start batched LLT for A_data
        int* info_array = static_cast<int*>(
                MemoryManager::Malloc(batch_size * sizeof(int), device));

        hipsolverStatus_t potrf_batched_status = potrf_cuda_batched<scalar_t>(
                handle, cols, A_array, cols, info_array, batch_size);


        OPEN3D_CUSOLVER_CHECK_WITH_INFO_ARRAY(
                potrf_batched_status, "potrfBatched failed in LLTCUDABatched",
                info_array, batch_size, device);

        MemoryManager::Free(info_array, device);
        MemoryManager::Free(A_array, device);
        // end batched LLT for A_data

        Blob dinfo(sizeof(int), device);
        scalar_t* B_ptr = static_cast<scalar_t*>(B_data);

        for (int i = 0; i < batch_size; i++) {
            OPEN3D_CUSOLVER_CHECK_WITH_DINFO(
                    potrs_cuda<scalar_t>(
                            handle, cols, cols,
                            A_ptr + (i * cols * cols), cols,
                            B_ptr + (i * cols * cols), cols,
                            static_cast<int*>(dinfo.GetDataPtr())),
                    "potrs failed in SolveCUDALLTBatched",
                    static_cast<int*>(dinfo.GetDataPtr()), device);
        }
    });
}

}  // namespace core
}  // namespace open3d
