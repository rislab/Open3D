#include "open3d/core/CUDAUtils.h"
#include "open3d/core/ParallelFor.h"
#include "open3d/core/linalg/LLTBatchedImpl.h"
#include "open3d/core/linalg/LapackWrapper.h"
#include "open3d/core/linalg/LinalgUtils.h"

namespace open3d {
namespace core {

void LLTBatchedCUDA(void* A_data,
                    int64_t batch_size,
                    int64_t cols,  // NOTE: this is a square matrix
                    Dtype dtype,
                    const Device& device) {
    hipsolverHandle_t handle = CuSolverContext::GetInstance()->GetHandle();
    DISPATCH_LINALG_DTYPE_TO_TEMPLATE(dtype, [&]() {
        scalar_t* output_ptr = static_cast<scalar_t*>(A_data);
        scalar_t** A_array = static_cast<scalar_t**>(
                MemoryManager::Malloc(batch_size * sizeof(scalar_t*), device));
        ParallelFor(device, batch_size,
                    [=] OPEN3D_DEVICE(int64_t workload_idx) {
                        A_array[workload_idx] =
                                output_ptr + (workload_idx * cols * cols);
                    });

        int* info_array = static_cast<int*>(
                MemoryManager::Malloc(batch_size * sizeof(int), device));

        hipsolverStatus_t potrf_batched_status = potrf_cuda_batched<scalar_t>(
                handle, cols, A_array, cols, info_array, batch_size);

        OPEN3D_CUSOLVER_CHECK_WITH_INFO_ARRAY(
                potrf_batched_status, "potrfBatched failed in LLTCUDABatched",
                info_array, batch_size, device);

        MemoryManager::Free(info_array, device);
        MemoryManager::Free(A_array, device);
    });
}

}  // namespace core
}  // namespace open3d
